#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2022, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_CLAMP_OPERATOR_INSTANTIATE
#include "lbann/operators/math/clamp.hpp"
#include "lbann/utils/gpu/sync_info_helpers.hpp"

namespace lbann {

namespace {

/** CUDA kernel for forward prop computation. */
template <typename DataT>
__global__ void fp_kernel(DataT min,
                          DataT max,
                          El::Int height,
                          El::Int width,
                          const DataT* __restrict__ input,
                          El::Int input_ldim,
                          DataT* __restrict__ output,
                          El::Int output_ldim)
{
  const El::Int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int size = height * width;
  const El::Int num_threads = blockDim.x * gridDim.x;
  for (El::Int pos = gid; pos < size; pos += num_threads) {
    const auto& row = pos % height;
    const auto& col = pos / height;
    const auto& x = input[row + col * input_ldim];
    auto& y = output[row + col * output_ldim];
    if (x <= min) {
      y = min;
    }
    else if (x >= max) {
      y = max;
    }
    else {
      y = x;
    }
  }
}

/** GPU kernel for backprop computation. */
template <typename DataT>
__global__ void bp_kernel(DataT min,
                          DataT max,
                          El::Int height,
                          El::Int width,
                          const DataT* __restrict__ input,
                          El::Int input_ldim,
                          const DataT* __restrict__ gradient_wrt_output,
                          El::Int gradient_wrt_output_ldim,
                          DataT* __restrict__ gradient_wrt_input,
                          El::Int gradient_wrt_input_ldim)
{
  const El::Int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const El::Int size = height * width;
  const El::Int num_threads = blockDim.x * gridDim.x;
  for (El::Int pos = gid; pos < size; pos += num_threads) {
    const auto& row = pos % height;
    const auto& col = pos / height;
    const auto& x = input[row + col * input_ldim];
    const auto& dy = gradient_wrt_output[row + col * gradient_wrt_output_ldim];
    auto& dx = gradient_wrt_input[row + col * gradient_wrt_input_ldim];
    dx = (x <= min || x >= max) ? DataT(0.f) : dy;
  }
}

/** Local forward prop computation. */
template <typename DataT>
void local_fp(DataT min,
              DataT max,
              El::Matrix<DataT, El::Device::GPU> const& input,
              El::Matrix<DataT, El::Device::GPU>& output)
{

  // Get CUDA grid dimensions
  // Note: Maximum CUDA grid dimension is 2^32-1
  // (https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications).
  // TODO: HIP/ROCM notes
  const El::Int height = input.Height();
  const El::Int width = input.Width();
  const El::Int block_dim = 256;
  El::Int grid_dim = (height * width + block_dim - 1) / block_dim;
  if (sizeof(El::Int) > sizeof(unsigned int) &&
      grid_dim > std::numeric_limits<uint32_t>::max()) {
    grid_dim = std::numeric_limits<uint32_t>::max();
  }

  // Launch GPU kernel
  if (grid_dim > 0) {
    auto multisync =
      El::MakeMultiSync(gpu::get_sync_info(input), gpu::get_sync_info(output));
    hydrogen::gpu::LaunchKernel(fp_kernel<DataT>,
                                grid_dim,
                                block_dim,
                                0,
                                multisync,
                                min,
                                max,
                                height,
                                width,
                                input.LockedBuffer(),
                                input.LDim(),
                                output.Buffer(),
                                output.LDim());
  }
}

/** Local backprop computation. */
template <typename DataT>
void local_bp(DataT min,
              DataT max,
              El::Matrix<DataT, El::Device::GPU> const& input,
              El::Matrix<DataT, El::Device::GPU> const& gradient_wrt_output,
              El::Matrix<DataT, El::Device::GPU>& gradient_wrt_input)
{

  // Get CUDA grid dimensions
  // Note: Maximum CUDA grid dimension is 2^32-1
  // (https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications).
  // TODO: HIP/ROCM notes
  const El::Int height = input.Height();
  const El::Int width = input.Width();
  const El::Int block_dim = 256;
  El::Int grid_dim = (height * width + block_dim - 1) / block_dim;
  if (sizeof(El::Int) > sizeof(unsigned int) &&
      grid_dim > std::numeric_limits<uint32_t>::max()) {
    grid_dim = std::numeric_limits<uint32_t>::max();
  }

  // Launch GPU kernel
  if (grid_dim > 0) {
    auto multisync = El::MakeMultiSync(gpu::get_sync_info(gradient_wrt_output),
                                       gpu::get_sync_info(gradient_wrt_input));
    hydrogen::gpu::LaunchKernel(bp_kernel<DataT>,
                                grid_dim,
                                block_dim,
                                0,
                                multisync,
                                min,
                                max,
                                height,
                                width,
                                input.LockedBuffer(),
                                input.LDim(),
                                gradient_wrt_output.LockedBuffer(),
                                gradient_wrt_output.LDim(),
                                gradient_wrt_input.Buffer(),
                                gradient_wrt_input.LDim());
  }
}

} // namespace

template <typename DataT, El::Device D>
void ClampOperator<DataT, D>::fp_compute_local(
  std::vector<ConstLocalInputTensorType> inputs,
  std::vector<LocalOutputTensorType> outputs) const
{
  LBANN_ASSERT(inputs.size() == 1 && outputs.size() == 1);
  local_fp(this->m_min, this->m_max, inputs[0].data(), outputs[0].data());
}

template <typename DataT, El::Device D>
void ClampOperator<DataT, D>::bp_compute_local(
  std::vector<ConstLocalInputTensorType> inputs,
  std::vector<ConstLocalOutputTensorType> gradient_wrt_outputs,
  std::vector<LocalInputTensorType> gradient_wrt_inputs) const
{
  LBANN_ASSERT(inputs.size() == 1 && gradient_wrt_outputs.size() == 1 &&
               gradient_wrt_inputs.size() == 1);
  local_bp(this->m_min,
           this->m_max,
           inputs[0].data(),
           gradient_wrt_outputs[0].data(),
           gradient_wrt_inputs[0].data());
}

#define PROTO(T) template class ClampOperator<T, El::Device::GPU>

#define LBANN_INSTANTIATE_GPU_HALF
#include "lbann/macros/instantiate.hpp"

} // namespace lbann

#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2022, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#include "lbann/utils/dim_helpers.hpp"
#include "lbann/utils/gpu/helpers.hpp"

#ifdef LBANN_HAS_CUDA

namespace lbann {
namespace gpu_lib {

// -------------------------------------------------------------
// Device properties
// -------------------------------------------------------------

dim3 max_grid_dims()
{
  static dim3 max_grid_dims_(0, 0, 0);
  if (max_grid_dims_.x == 0) {
    int device = 0;
    hipDeviceProp_t prop;
    CHECK_CUDA(hipGetDevice(&device));
    CHECK_CUDA(hipGetDeviceProperties(&prop, device));
    max_grid_dims_.x = prop.maxGridSize[0];
    max_grid_dims_.y = prop.maxGridSize[1];
    max_grid_dims_.z = prop.maxGridSize[2];
    if (max_grid_dims_.x == 0) {
      LBANN_ERROR("Could not setup max CUDA grid size");
    }
  }
  return max_grid_dims_;
}

} // namespace gpu_lib
} // namespace lbann

namespace lbann {
namespace cuda {

// -------------------------------------------------------------
// event_wrapper
// -------------------------------------------------------------

event_wrapper::event_wrapper() : m_event(nullptr), m_stream(0)
{
  CHECK_CUDA(hipEventCreateWithFlags(&m_event, hipEventDisableTiming));
}

event_wrapper::event_wrapper(const event_wrapper& other)
  : m_event(nullptr), m_stream(other.m_stream)
{
  CHECK_CUDA(hipEventCreateWithFlags(&m_event, hipEventDisableTiming));
  if (!other.query()) {
    record(m_stream);
  }
}

event_wrapper& event_wrapper::operator=(const event_wrapper& other)
{
  m_stream = other.m_stream;
  if (!other.query()) {
    record(m_stream);
  }
  return *this;
}

event_wrapper::~event_wrapper() { hipEventDestroy(m_event); }

void event_wrapper::record(hipStream_t stream)
{
  m_stream = stream;
  CHECK_CUDA(hipEventRecord(m_event, m_stream));
}

bool event_wrapper::query() const
{
  const auto& status = hipEventQuery(m_event);
  switch (status) {
  case hipSuccess:
    return true;
  case hipErrorNotReady:
    return false;
  default:
    CHECK_CUDA(status);
    return false;
  }
}

void event_wrapper::synchronize() { CHECK_CUDA(hipEventSynchronize(m_event)); }

hipEvent_t& event_wrapper::get_event() { return m_event; }

// -----------------------------
// Graph
// -----------------------------

Graph::Graph(hipGraph_t graph) : graph_{graph} {}

Graph::~Graph()
{
  if (graph_) {
    // Don't check status to avoid exceptions
    hipGraphDestroy(graph_);
  }
}

Graph::Graph(const Graph& other)
{
  if (other.graph_) {
    CHECK_CUDA(hipGraphClone(&graph_, other.graph_));
  }
}

Graph::Graph(Graph&& other) : graph_{other.graph_} { other.graph_ = nullptr; }

Graph& Graph::operator=(Graph other)
{
  swap(other, *this);
  return *this;
}

void swap(Graph& first, Graph& second)
{
  std::swap(first.graph_, second.graph_);
}

void Graph::reset(hipGraph_t graph)
{
  if (graph_) {
    CHECK_CUDA(hipGraphDestroy(graph_));
  }
  graph_ = graph;
}

hipGraph_t Graph::release()
{
  auto old_graph = graph_;
  graph_ = nullptr;
  return old_graph;
}

hipGraph_t Graph::get() const noexcept { return graph_; }

Graph::operator hipGraph_t() const noexcept { return get(); }

void Graph::create()
{
  if (!graph_) {
    CHECK_CUDA(hipGraphCreate(&graph_, 0));
  }
}

void Graph::begin_capture(hipStream_t stream, hipStreamCaptureMode mode)
{

  // Check that stream is valid
  // Note (tym 9/22/20): As of CUDA 11.0.3, support for stream capture
  // on default stream is not supported.
  if (stream == 0) {
    LBANN_ERROR("attempting to capture default CUDA stream");
  }

  // Check whether CUDA stream is already being captured
  hipStreamCaptureStatus capture_status;
  CHECK_CUDA(hipStreamIsCapturing(stream, &capture_status));
  switch (capture_status) {
  case hipStreamCaptureStatusNone:
    break;
  case hipStreamCaptureStatusActive:
    LBANN_ERROR("CUDA stream is already being captured");
    break;
  case hipStreamCaptureStatusInvalidated: {
    hipGraph_t graph;
    CHECK_CUDA(hipStreamEndCapture(stream, &graph));
    Graph temp(graph);
  } break;
  default:
    LBANN_ERROR("unrecognized status for CUDA stream capture ",
                "(",
                static_cast<int>(capture_status),
                ")");
  }

  // Start capturing CUDA stream
  CHECK_CUDA(hipStreamBeginCapture(stream, mode));
}

Graph Graph::end_capture(hipStream_t stream)
{

  // Check whether CUDA stream is already being captured
  hipStreamCaptureStatus capture_status;
  CHECK_CUDA(hipStreamIsCapturing(stream, &capture_status));
  switch (capture_status) {
  case hipStreamCaptureStatusNone:
    LBANN_ERROR("CUDA stream is not being captured");
    break;
  case hipStreamCaptureStatusActive:
    break;
  case hipStreamCaptureStatusInvalidated: {
    hipGraph_t graph;
    CHECK_CUDA(hipStreamEndCapture(stream, &graph));
    Graph temp(graph);
    LBANN_ERROR("CUDA stream capture has failed");
  } break;
  default:
    LBANN_ERROR("unrecognized status for CUDA stream capture ",
                "(",
                static_cast<int>(capture_status),
                ")");
  }

  // Finish capturing CUDA stream
  hipGraph_t graph;
  CHECK_CUDA(hipStreamEndCapture(stream, &graph));
  return Graph(graph);
}

// -----------------------------
// ExecutableGraph
// -----------------------------

ExecutableGraph::ExecutableGraph(hipGraphExec_t graph_exec)
  : graph_exec_{graph_exec}
{}

ExecutableGraph::ExecutableGraph(hipGraph_t graph)
{
  if (!graph) {
    LBANN_ERROR(
      "attempted to instantiate hipGraphExec_t from null hipGraph_t object");
  }
  constexpr size_t log_size = BUFSIZ;
  char log_buffer[log_size];
  const auto status =
    hipGraphInstantiate(&graph_exec_, graph, nullptr, log_buffer, log_size);
  if (status != hipSuccess && log_buffer[0] != '\0') {
    log_buffer[log_size - 1] = '\0';
    LBANN_WARNING(log_buffer);
  }
  CHECK_CUDA(status);
}

ExecutableGraph::~ExecutableGraph()
{
  if (graph_exec_) {
    // Don't check status to avoid exceptions
    hipGraphExecDestroy(graph_exec_);
  }
}

ExecutableGraph::ExecutableGraph(ExecutableGraph&& other)
  : graph_exec_{other.graph_exec_}
{
  other.graph_exec_ = nullptr;
}

ExecutableGraph& ExecutableGraph::operator=(ExecutableGraph other)
{
  swap(other, *this);
  return *this;
}

void swap(ExecutableGraph& first, ExecutableGraph& second)
{
  std::swap(first.graph_exec_, second.graph_exec_);
}

void ExecutableGraph::reset(hipGraphExec_t graph_exec)
{
  if (graph_exec_) {
    CHECK_CUDA(hipGraphExecDestroy(graph_exec_));
  }
  graph_exec_ = graph_exec;
}

hipGraphExec_t ExecutableGraph::release()
{
  auto old_graph_exec = graph_exec_;
  graph_exec_ = nullptr;
  return old_graph_exec;
}

hipGraphExec_t ExecutableGraph::get() const noexcept { return graph_exec_; }

ExecutableGraph::operator hipGraphExec_t() const noexcept { return get(); }

void ExecutableGraph::launch(hipStream_t stream) const
{
  if (!graph_exec_) {
    LBANN_ERROR("attempted to launch null hipGraphExec_t");
  }
  CHECK_CUDA(hipGraphLaunch(graph_exec_, stream));
}

void ExecutableGraph::update(hipGraph_t graph)
{

  // Make sure CUDA graph is valid
  if (!graph) {
    LBANN_ERROR("attempting to update hipGraphExec_t with null hipGraph_t");
  }

  // Try updating executable CUDA graph
#if (__CUDACC_VER_MAJOR__ * 100 + __CUDACC_VER_MINOR__) < 1002 // < 10.2
  reset();
#else  // >= 10.2
  if (graph_exec_) {
    hipGraphNode_t error_node;
    hipGraphExecUpdateResult result;
    auto status = hipGraphExecUpdate(graph_exec_, graph, &error_node, &result);
    switch (status) {
    case hipSuccess:
      break;
    case hipErrorGraphExecUpdateFailure:
      reset();
      break;
    default:
      CHECK_CUDA(status);
      reset();
    }
  }
#endif // CUDA version >= 10.02

  // If update failed, create new executable CUDA graph
  if (!graph_exec_) {
    *this = ExecutableGraph(graph);
  }
}

// -------------------------------------------------------------
// Helper functions for tensor operations
// -------------------------------------------------------------

namespace {

using int4 = gpu_lib::array<int, 4>;

/**
 *  Block dimensions: bdimx x bdimy x bdimz
 *
 *  Grid dimensions: (dim[3] / bdimx) x (dim[2] / bdimy) x (dim[1] / bdimx)
 */
template <typename TensorDataType>
__global__ void copy_4d_kernel(int4 dims,
                               const TensorDataType* __restrict__ input,
                               int4 input_strides,
                               TensorDataType* __restrict__ output,
                               int4 output_strides)
{

  // Indices
  const auto& gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const auto& gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const auto& gidz = threadIdx.z + blockIdx.z * blockDim.z;
  const auto& nthreadsx = gridDim.x * blockDim.x;
  const auto& nthreadsy = gridDim.y * blockDim.y;
  const auto& nthreadsz = gridDim.z * blockDim.z;

  for (int i0 = 0; i0 < dims[0]; ++i0) {
    for (int i1 = gidz; i1 < dims[1]; i1 += nthreadsz) {
      for (int i2 = gidy; i2 < dims[2]; i2 += nthreadsy) {
        for (int i3 = gidx; i3 < dims[3]; i3 += nthreadsx) {
          const auto& x = input[i0 * input_strides[0] + i1 * input_strides[1] +
                                i2 * input_strides[2] + i3 * input_strides[3]];
          auto& y = output[i0 * output_strides[0] + i1 * output_strides[1] +
                           i2 * output_strides[2] + i3 * output_strides[3]];
          y = x;
        }
      }
    }
  }
}

} // namespace

template <typename TensorDataType>
void copy_tensor(hipStream_t stream,
                 const std::vector<size_t>& dims,
                 const TensorDataType* input,
                 const std::vector<size_t>& input_strides,
                 TensorDataType* output,
                 const std::vector<size_t>& output_strides)
{

  // Check inputs
  if (dims.empty() || dims.size() > 4) {
    LBANN_ERROR("invalid number of tensor dimensions (", dims.size(), ")");
  }
  if (dims.size() != input_strides.size()) {
    LBANN_ERROR("number of input strides (",
                input_strides.size(),
                ") ",
                "does not match number of tensor dimensions (",
                dims.size(),
                ")");
  }
  if (dims.size() != output_strides.size()) {
    LBANN_ERROR("number of output strides (",
                output_strides.size(),
                ") ",
                "does not match number of tensor dimensions (",
                dims.size(),
                ")");
  }

  // Pad tensor dimensions to 4D
  std::vector<int> rdims(dims.rbegin(), dims.rend()),
    input_rstrides(input_strides.rbegin(), input_strides.rend()),
    output_rstrides(output_strides.rbegin(), output_strides.rend());
  rdims.resize(4, 1);
  input_rstrides.resize(4, input_rstrides.back());
  output_rstrides.resize(4, output_rstrides.back());

  // Launch CUDA kernel
  const auto size = get_linear_size(dims);
  if (size > 0) {
    constexpr size_t block_size = 64;
    dim3 block_dims, grid_dims;
    block_dims.x = block_size;
    block_dims.y = 1;
    block_dims.z = 1;
    grid_dims.x = (rdims[0] + block_dims.x - 1) / block_dims.x;
    grid_dims.y = (rdims[1] + block_dims.y - 1) / block_dims.y;
    grid_dims.z = (rdims[2] + block_dims.z - 1) / block_dims.z;
    grid_dims.y = El::Min(grid_dims.y, 65535);
    grid_dims.z = El::Min(grid_dims.z, 65535);
    copy_4d_kernel<<<grid_dims, block_dims, 0, stream>>>(
      {rdims[3], rdims[2], rdims[1], rdims[0]},
      input,
      {input_rstrides[3],
       input_rstrides[2],
       input_rstrides[1],
       input_rstrides[0]},
      output,
      {output_rstrides[3],
       output_rstrides[2],
       output_rstrides[1],
       output_rstrides[0]});
  }
}

#if defined(LBANN_HAS_HALF) && defined(LBANN_HAS_GPU_HALF)
template <>
void copy_tensor<cpu_fp16>(hipStream_t stream,
                           const std::vector<size_t>& dims,
                           const cpu_fp16* input,
                           const std::vector<size_t>& input_strides,
                           cpu_fp16* output,
                           const std::vector<size_t>& output_strides)
{
  copy_tensor<fp16>(stream,
                    dims,
                    reinterpret_cast<const fp16*>(input),
                    input_strides,
                    reinterpret_cast<fp16*>(output),
                    output_strides);
}
#endif // defined(LBANN_HAS_HALF) && defined(LBANN_HAS_GPU_HALF)

// Explicit template instantiation
#define PROTO(T)                                                               \
  template void copy_tensor<T>(hipStream_t stream,                            \
                               const std::vector<size_t>& dims,                \
                               const T* input,                                 \
                               const std::vector<size_t>& input_strides,       \
                               T* output,                                      \
                               const std::vector<size_t>& output_strides);
#define LBANN_INSTANTIATE_GPU_HALF
#define LBANN_INSTANTIATE_CPU_HALF
#include "lbann/macros/instantiate.hpp"
#undef PROTO

void mem_copy_async(void* output,
                    const void* input,
                    const size_t count,
                    hipMemcpyKind kind,
                    hipStream_t stream)
{
  CHECK_CUDA(hipMemcpyAsync(output, input, count, kind, stream));
}

} // namespace cuda
} // namespace lbann

#endif // LBANN_HAS_CUDA
